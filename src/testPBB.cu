#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostSkel.cu.h"

// R seems to be max value of the elements of the array
void initArray(int32_t* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i] = (rand() % M) - R;
    }
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 */ 
int bandwidthMemcpy( const uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                   , const size_t   N     // length of the input array
                   , int* d_in            // device input  of length N
                   , int* d_out           // device result of length N
) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + B - 1) / B;
    naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int) * 1.0e-3f / elapsed;
        printf("Naive Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n\n\n"
              , elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
    return 0;
}


int scanIncAddI32( const uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                 , const size_t   N     // length of the input array
                 , int* h_in            // host input    of size: N * sizeof(int)
                 , int* d_in            // device input  of size: N * sizeof(ElTp)
                 , int* d_out           // device result of size: N * sizeof(int)
) {
    const size_t mem_size = N * sizeof(int);
    int* d_tmp;
    int* h_out = (int*)malloc(mem_size);
    int* h_ref = (int*)malloc(mem_size);
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(int));
    hipMemset(d_out, 0, N*sizeof(int));

    // dry run to exercise d_tmp allocation
    scanInc< Add<int> > ( B, N, d_out, d_in, d_tmp );

    // time the GPU computation
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int i=0; i<RUNS_GPU; i++) {
        scanInc< Add<int> > ( B, N, d_out, d_in, d_tmp );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
    double gigaBytesPerSec = N  * (2*sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
    printf("Scan Inclusive AddI32 GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"
          , elapsed, gigaBytesPerSec);

    gpuAssert( hipPeekAtLastError() );

    { // sequential computation
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_CPU; i++) {
            int acc = 0;
            for(uint32_t i=0; i<N; i++) {
                acc += h_in[i];
                h_ref[i] = acc;
            }
        }
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_CPU;
        double gigaBytesPerSec = N * (sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
        printf("Scan Inclusive AddI32 CPU Sequential runs in: %lu microsecs, GB/sec: %.2f\n"
              , elapsed, gigaBytesPerSec);
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: Scan Inclusive AddI32 at index %d, dev-val: %d, host-val: %d\n"
                      , i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("Scan Inclusive AddI32: VALID result!\n\n");
    }

    free(h_out);
    free(h_ref);
    hipFree(d_tmp);
    
    return 0;
}


int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <array-length> <block-size>\n", argv[0]);
        exit(1);
    }

    initHwd();

    const uint32_t N = atoi(argv[1]);
    const uint32_t B = atoi(argv[2]);

    printf("Testing parallel basic blocks for input length: %d and CUDA-block size: %d\n\n\n", N, B);

    const size_t mem_size = N*sizeof(int);
    int* h_in    = (int*) malloc(mem_size);
    int* d_in;
    int* d_out;
    hipMalloc((void**)&d_in ,   mem_size);
    hipMalloc((void**)&d_out,   mem_size);

    initArray(h_in, N, 13);
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);
 
    // computing a "realistic/achievable" bandwidth figure
    bandwidthMemcpy(B, N, d_in, d_out);

    { // inclusive scan with int addition
        scanIncAddI32   (B, N, h_in, d_in, d_out);
    }


    // cleanup memory
    free(h_in);
    hipFree(d_in );
    hipFree(d_out);
}
