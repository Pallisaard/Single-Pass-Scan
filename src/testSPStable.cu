#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostSkel.cu.h"

#ifndef INCLUDE_QUAD
#define INCLUDE_QUAD 0
#endif

// Initialize an array of int32_t with random values between -R and R.
// Array has length N.
// R seems to be max value of the elements of the array.
void initArrayInt32(int32_t* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i] = (rand() % M) - R;
    }
}

// Initialize an array of Quad<int32_t> with random values between -R and R.
// Array has length N.
// R seems to be max value of the elements of the array.
void initArrayQuadInt32(Quad<int32_t>* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i].x = (rand() % M) - R;
        inp_arr[i].y = (rand() % M) - R;
        inp_arr[i].z = (rand() % M) - R;
        inp_arr[i].w = (rand() % M) - R;
    }
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */ 
template<typename T>
int bandwidthCudaMemcpy(const size_t N, T* d_in, T* d_out) {
    // dry run to exercise the d_out allocation!
    const size_t mem_size = N * sizeof(T);
	hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
			hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(T) * 1.0e-3f / elapsed;
        printf("%.2f,", gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );

    return 0;
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */
 template<typename T>
int bandwidthMemcpy(const size_t N, T* d_in, T* d_out) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + 1024 - 1) / 1024;
    naiveMemcpy<T><<< num_blocks, 1024>>>(d_out, d_in, N);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<T><<< num_blocks, 1024 >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(T) * 1.0e-3f / elapsed;
        printf("%.2f,", gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );

    return 0;
}

/*
 * singlePassScanAuxBlock performs a single pass scan using an auxiliary block.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int singlePassScanAuxBlock(const size_t N, T* h_in,
	                       T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
    hipMemset(d_out, 0, N*sizeof(T));

    uint32_t num_blocks = (N+B*Q-1)/(B*Q) + 1;  // We add 1 to be our auxiliary block.
	size_t f_array_size = num_blocks - 1;
    int32_t* IDAddr;
    uint32_t* flagArr;
    T* aggrArr;
    T* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, -1, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(T));
    hipMemset(aggrArr, 0, f_array_size * sizeof(T));
    hipMalloc(&prefixArr, f_array_size * sizeof(T));
    hipMemset(prefixArr, 0, f_array_size * sizeof(T));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel1<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
	// time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, -1, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0, f_array_size * sizeof(T));
            hipMemset(prefixArr, 0, f_array_size * sizeof(T));
            SinglePassScanKernel1<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
            // printf("gpu %d\n", i + 1);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
        printf("%.2f,", gigaBytesPerSec);
    }
    gpuAssert( hipPeekAtLastError() );

	free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);

    return 0;
}

/*
 * singlePassScanLookback performs a single pass scan using lookback.
 * N - length of the input array
 * h_in - host input    of size: N * sizeof(int)
 * d_in - device input  of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int singlePassScanLookback(const size_t N, T* h_in,
                                T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
	hipMemset(d_out, 0,  N *sizeof(T));

	uint32_t num_blocks = (N+B*Q-1)/(B*Q);
    size_t f_array_size = num_blocks;
    int32_t* IDAddr;
    uint32_t* flagArr;
    T* aggrArr;
    T* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, 0, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(T));
    hipMemset(aggrArr, 0, f_array_size * sizeof(T));
    hipMalloc(&prefixArr, f_array_size * sizeof(T));
    hipMemset(prefixArr, 0, f_array_size * sizeof(T));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel2<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    // time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, 0, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0, f_array_size * sizeof(T));
            hipMemset(prefixArr, 0, f_array_size * sizeof(T));
            SinglePassScanKernel2<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
        printf("%.2f,", gigaBytesPerSec);
    }
    gpuAssert( hipPeekAtLastError() );

	free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);

    return 0;
}

/*
 * cpuSeqScan computes a sequencial scan on the cpu.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int cpuSeqScan(const size_t N, T* h_in,
	           T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);

    // dry run to exercise the h_out allocation!
    hipDeviceSynchronize();
    T acc = T();
    for(uint32_t i=0; i<N; i++) {
        acc = h_in[i] + acc;
        h_out[i] = acc;
    }
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    // The CPU we might as well just add the benchmark
    { // sequential computation
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_CPU; i++) {
            T acc = T();
            for(uint32_t i=0; i<N; i++) {
                acc = h_in[i] + acc;
                h_out[i] = acc;
            }
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_CPU;
        double gigaBytesPerSec = N * 2 * sizeof(T) * 1.0e-3f / elapsed;
        printf("%.2f,", gigaBytesPerSec);
    }

    free(h_out);
    return 0;
}

/*
 * scanIncAddI32 computes a scan inclusive add on the GPU using the
 * scanInc kernel we implemented in assignment 2.
 * b_size - desired CUDA block size ( <= 1024, multiple of 32)
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 */
template<typename T>
int scanIncAdd(const uint32_t b_size, const size_t N, T* h_in,
				  T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* d_tmp;
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(T));
    hipMemset(d_out, 0, N*sizeof(T));

    // dry run to exercise d_tmp allocation
    scanInc< Add<T> > ( b_size, N, d_out, d_in, d_tmp );

    // time the GPU computation
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int i=0; i<RUNS_GPU; i++) {
        scanInc< Add<T> > ( b_size, N, d_out, d_in, d_tmp );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
    double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
	printf("%.2f", gigaBytesPerSec);

    gpuAssert( hipPeekAtLastError() );

    free(h_out);
    free(h_ref);
    hipFree(d_tmp);

    return 0;
}

int i32Experiments(const uint32_t N) {
    const size_t mem_size = N*sizeof(int32_t);
    int32_t* h_in    = (int32_t*) malloc(mem_size);
    int32_t* d_in;
    int32_t* d_out;
    hipMalloc((void**)&d_in ,   mem_size);
    hipMalloc((void**)&d_out,   mem_size);

    initArrayInt32(h_in, N, 13);
	hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // Scan experiments.
    {
		// computing a "realistic/achievable" bandwidth figure
		bandwidthCudaMemcpy<int32_t>(N, d_in, d_out);
		bandwidthMemcpy<int32_t>(N, d_in, d_out);
		// Scan experiments.
        cpuSeqScan<int32_t>(N, h_in, d_in, d_out);
        singlePassScanAuxBlock<int32_t>(N, h_in, d_in, d_out);
        singlePassScanLookback<int32_t>(N, h_in, d_in, d_out);
		if (N != 2 << 21)
			scanIncAdd<int32_t>(B, N, h_in, d_in, d_out);
    }

    // cleanup memory
    free(h_in);
    hipFree(d_in);
    hipFree(d_out);

	return 0;
}

#if INCLUDE_QUAD == 1
int quadInt32Experiments(const uint32_t N) {
	const size_t mem_size = N*sizeof(Quad<int32_t>);
	Quad<int32_t>* h_in    = (Quad<int32_t>*) malloc(mem_size);
	Quad<int32_t>* d_in;
	Quad<int32_t>* d_out;
	hipMalloc((void**)&d_in ,   mem_size);
	hipMalloc((void**)&d_out,   mem_size);

	initArrayQuadInt32(h_in, N, 13);
	hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

	// Scan experiments.
	{
		// computing a "realistic/achievable" bandwidth figure
		bandwidthCudaMemcpy<Quad<int32_t>>(N, d_in, d_out);
		bandwidthMemcpy<Quad<int32_t>>(N, d_in, d_out);
		// bandwidthGlgShrMemcpyInt32(N, h_in, d_in, d_out);
		// Scan experiments.
		cpuSeqScan<Quad<int32_t>>(N, h_in, d_in, d_out);
		singlePassScanAuxBlock<Quad<int32_t>>(N, h_in, d_in, d_out);
		singlePassScanLookback<Quad<int32_t>>(N, h_in, d_in, d_out);
		if (N != 2 << 19)
			scanIncAdd<Quad<int32_t>>(B, N, h_in, d_in, d_out);
	}

	// cleanup memory
	free(h_in);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
#endif

int main (int argc, char * argv[]) {
    if (argc != 1) {
        printf("Usage: %s\n", argv[0]);
        exit(1);
    }

    initHwd();

    printf("Testing parallel basic blocks for CUDA-block size: %d and Q: %d\n\n\n", B, Q);

	printf("N (2^i),hipMemcpy,naiveMemcpy,cpuSeqScan,singlePassScanAuxBlock,singlePassScanLookback,scanIncAdd\n");
	for (uint32_t i = 10; i < 31; i++) {
		printf("%d,", i);
		uint32_t N = 2 << i;
		// printf("N: %d,", N);
		i32Experiments(N);
		printf("\n");
	}
	
	#if INCLUDE_QUAD
	printf("N,hipMemcpy,naiveMemcpy,cpuSeqScan,singlePassScanAuxBlock,singlePassScanLookback,scanIncAdd\n");
	for (uint32_t i = 10; i < 30; i++) {
		printf("%d,", i);
		uint32_t N = 2 << i;
		quadInt32Experiments(N);
		printf("\n");
	}
	#endif

	return 0;
}
