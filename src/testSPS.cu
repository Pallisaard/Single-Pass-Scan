#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostSkel.cu.h"


// R seems to be max value of the elements of the array
void initArray(int32_t* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i] = (rand() % M) - R;
    }
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */ 
int bandwidthCudaMemcpy(const size_t N, int* d_in, int* d_out) {
    // dry run to exercise the d_out allocation!
    const size_t mem_size = N * sizeof(int32_t);
	hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
			hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int32_t) * 1.0e-3f / elapsed;
        printf("hipMemcpy runs in: %lu microsecs, GB/sec: %.2f\n",
			   elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
	printf("\n");
    return 0;
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */ 
int bandwidthMemcpy(const size_t N, int* d_in, int* d_out) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + 1024 - 1) / 1024;
    naiveMemcpy<<< num_blocks, 1024 >>>(d_out, d_in, N);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<<< num_blocks, 1024 >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int32_t) * 1.0e-3f / elapsed;
        printf("Naive Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"	,
		       elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
	printf("\n");
    return 0;
}

/*
 * Measure a memcpy like kernel using registers in the GPU akin to
 * what the other methods use.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 */ 
 int bandwidthRegMemcpy(const size_t N, int* h_in, int* d_in, int* d_out) {
    const size_t mem_size = N * sizeof(int32_t);
	int32_t* h_out = (int32_t*)malloc(mem_size);
    int32_t* h_ref = (int32_t*)malloc(mem_size);

    hipMemset(d_out, 0, N*sizeof(int32_t));

	// dry run to exercise the d_out allocation!
	const uint32_t num_blocks = (N + B * Q - 1) / (B * Q);
	regMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
	hipDeviceSynchronize();

	double gigaBytesPerSec;
	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;

	{ // timing the GPU implementations
		gettimeofday(&t_start, NULL); 

		for(int i=0; i<RUNS_GPU; i++) {
			regMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
		}
		hipDeviceSynchronize();

		gettimeofday(&t_end, NULL);
		timeval_subtract(&t_diff, &t_end, &t_start);
		elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
		gigaBytesPerSec = 2 * N * sizeof(int32_t) * 1.0e-3f / elapsed;
		printf("Register Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n",
			   elapsed, gigaBytesPerSec);
	}

	gpuAssert( hipPeekAtLastError() );
	// The CPU we might as well just add the benchmark
    { // sequential computation
        for(uint32_t i=0; i<N; i++) {
            h_ref[i] = h_in[i];
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: register memory at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("register memcpy: VALID result!\n");
    }
	printf("\n");

    free(h_out);
    free(h_ref);

	return 0;
}

/*
 * Measure a memcpy like kernel by copying from global to shared and then back
 * to global memory.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 */
int bandwidthGlgShrMemcpy(const size_t N, int* h_in, int* d_in, int* d_out) {
    const size_t mem_size = N * sizeof(int32_t);
	int32_t* h_out = (int32_t*)malloc(mem_size);
    int32_t* h_ref = (int32_t*)malloc(mem_size);

    hipMemset(d_out, 0, N*sizeof(int32_t));

	// dry run to exercise the d_out allocation!
	const uint32_t num_blocks = (N + B * Q - 1) / (B * Q);
	glbShrMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
	hipDeviceSynchronize();

	double gigaBytesPerSec;
	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;

	{ // timing the GPU implementations
		gettimeofday(&t_start, NULL); 

		for(int i=0; i<RUNS_GPU; i++) {
			glbShrMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
		}
		hipDeviceSynchronize();

		gettimeofday(&t_end, NULL);
		timeval_subtract(&t_diff, &t_end, &t_start);
		elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
		gigaBytesPerSec = 2 * N * sizeof(int32_t) * 1.0e-3f / elapsed;
		printf("glbShr Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"	
		, elapsed, gigaBytesPerSec);
	}

	gpuAssert( hipPeekAtLastError() );
	// The CPU we might as well just add the benchmark
    { // sequential computation
        for(uint32_t i=0; i<N; i++) {
            h_ref[i] = h_in[i];
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: register memory at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("register memcpy: VALID result!\n");
    }
	printf("\n");

    free(h_out);
    free(h_ref);

	return 0;
}

/*
 * singlePassScanAuxBlock performs a single pass scan using an auxiliary block.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
int singlePassScanAuxBlock(const size_t N, int32_t* h_in, int32_t* d_in, int32_t* d_out) {
    const size_t mem_size = N * sizeof(int32_t);
    int32_t* h_out = (int32_t*)malloc(mem_size);
    int32_t* h_ref = (int32_t*)malloc(mem_size);
    hipMemset(d_out, 0, N*sizeof(int32_t));

    uint32_t num_blocks = (N+B*Q-1)/(B*Q) + 1;  // We add 1 to be our auxiliary block.
	size_t f_array_size = num_blocks - 1;
    int32_t* IDAddr;
    uint32_t* flagArr;
    int32_t* aggrArr;
    int32_t* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, -1, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(int32_t));
    hipMemset(aggrArr, 0, f_array_size * sizeof(int32_t));
    hipMalloc(&prefixArr, f_array_size * sizeof(uint32_t));
    hipMemset(prefixArr, 0, f_array_size * sizeof(uint32_t));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel1<<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
	// time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, -1, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0, f_array_size * sizeof(int32_t));
            hipMemset(prefixArr, 0, f_array_size * sizeof(uint32_t));
            SinglePassScanKernel1<<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
            // printf("gpu %d\n", i + 1);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(int32_t) * 1.0e-3f / elapsed;
        printf("Single Pass Scan using aux block runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }
    gpuAssert( hipPeekAtLastError() );

    // The CPU we might as well just add the benchmark
    { // sequential computation
        int acc = 0;
        for(uint32_t i=0; i<N; i++) {
            acc = h_in[i] + acc;
            h_ref[i] = acc;
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: Single Pass Scan at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("Single pass scan using aux block: VALID result!\n");
    }
    free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);
	printf("\n");
    return 0;
}

/*
 * singlePassScanLookback performs a single pass scan using lookback.
 * N - length of the input array
 * h_in - host input    of size: N * sizeof(int)
 * d_in - device input  of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
int singlePassScanLookback(const size_t N, int32_t* h_in,
                           int32_t* d_in, int32_t* d_out) {
    const size_t mem_size = N * sizeof(int32_t);
    int32_t* h_out = (int32_t*)malloc(mem_size);
    int32_t* h_ref = (int32_t*)malloc(mem_size);
	hipMemset(d_out, 0,  N *sizeof(int32_t));

	uint32_t num_blocks = (N+B*Q-1)/(B*Q);
    size_t f_array_size = num_blocks;
    int32_t* IDAddr;
    uint32_t* flagArr;
    int32_t* aggrArr;
    int32_t* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, 0, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(int32_t));
    hipMemset(aggrArr, 0, f_array_size * sizeof(int32_t));
    hipMalloc(&prefixArr, f_array_size * sizeof(uint32_t));
    hipMemset(prefixArr, 0, f_array_size * sizeof(uint32_t));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel2<<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    // time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, 0, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0, f_array_size * sizeof(int32_t));
            hipMemset(prefixArr, 0, f_array_size * sizeof(uint32_t));
            SinglePassScanKernel2<<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(int32_t) * 1.0e-3f / elapsed;
        printf("Single Pass Scan using lookback runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }
    gpuAssert( hipPeekAtLastError() );

    // The CPU we might as well just add the benchmark
    { // sequential computation
        int acc = 0;
        for(uint32_t i=0; i<N; i++) {
            acc = h_in[i] + acc;
            h_ref[i] = acc;
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: Single Pass Scan at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("Single pass scan using lookback: VALID result!\n");
    }
    free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);
	printf("\n");
    return 0;
}

/*
 * cpuSeqScan computes a sequencial scan on the cpu.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
int cpuSeqScan(const size_t N, int32_t* h_in, int32_t* d_in, int32_t* d_out){
    const size_t mem_size = N * sizeof(int32_t);
    int32_t* h_out = (int32_t*)malloc(mem_size);

    // dry run to exercise the h_out allocation!
    hipDeviceSynchronize();
    int acc = 0;
    for(uint32_t i=0; i<N; i++) {
        acc = h_in[i] + acc;
        h_out[i] = acc;
    }
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    // The CPU we might as well just add the benchmark
    { // sequential computation
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_CPU; i++) {
            int acc = 0;
            for(uint32_t i=0; i<N; i++) {
                acc = h_in[i] + acc;
                h_out[i] = acc;
            }
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_CPU;
        double gigaBytesPerSec = N * (sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
        printf("Scan Inclusive AddI32 CPU Sequential runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }

    free(h_out);
	printf("\n");
    return 0;
}

/*
 * scanIncAddI32 computes a scan inclusive add on the GPU using the
 * scanInc kernel we implemented in assignment 2.
 * b_size - desired CUDA block size ( <= 1024, multiple of 32)
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 */
int scanIncAddI32(const uint32_t b_size, const size_t N, int* h_in,
				  int* d_in, int* d_out) {
    const size_t mem_size = N * sizeof(int);
    int* d_tmp;
    int* h_out = (int*)malloc(mem_size);
    int* h_ref = (int*)malloc(mem_size);
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(int));
    hipMemset(d_out, 0, N*sizeof(int));

    // dry run to exercise d_tmp allocation
    scanInc< Add<int> > ( b_size, N, d_out, d_in, d_tmp );

    // time the GPU computation
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int i=0; i<RUNS_GPU; i++) {
        scanInc< Add<int> > ( b_size, N, d_out, d_in, d_tmp );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
    double gigaBytesPerSec = N  * (2*sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
    printf("Scan Inclusive AddI32 GPU11 Kernel runs in: %lu microsecs, GB/sec: %.2f\n"
          , elapsed, gigaBytesPerSec);

    gpuAssert( hipPeekAtLastError() );

    { // sequential computation
		int acc = 0;
		for(uint32_t i=0; i<N; i++) {
			acc += h_in[i];
			h_ref[i] = acc;
		}
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: Scan Inclusive AddI32 at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("Scan Inclusive AddI32: VALID result!\n\n");
    }

    free(h_out);
    free(h_ref);
    hipFree(d_tmp);

    return 0;
}


int main (int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array-length>\n", argv[0]);
        exit(1);
    }

    initHwd();

    const uint32_t N = atoi(argv[1]);

    printf("Testing parallel basic blocks for input length: %d and CUDA-block size: %d and Q: %d\n\n\n", N, B, Q);

    const size_t mem_size = N*sizeof(int32_t);
    int32_t* h_in    = (int32_t*) malloc(mem_size);
    int32_t* d_in;
    int32_t* d_out;
    hipMalloc((void**)&d_in ,   mem_size);
    hipMalloc((void**)&d_out,   mem_size);

    initArray(h_in, N, 13);
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // computing a "realistic/achievable" bandwidth figure
	bandwidthMemcpy(N, d_in, d_out);
	bandwidthCudaMemcpy(N, d_in, d_out);
	bandwidthRegMemcpy(N, h_in, d_in, d_out);
	bandwidthGlgShrMemcpy(N, h_in, d_in, d_out);

    // Scan experiments.
    {
        singlePassScanLookback(N, h_in, d_in, d_out);
        singlePassScanAuxBlock(N, h_in, d_in, d_out);
		scanIncAddI32(B, N, h_in, d_in, d_out);
        cpuSeqScan(N, h_in, d_in, d_out);
    }

    // cleanup memory
    free(h_in);
    hipFree(d_in);
    hipFree(d_out);
}
