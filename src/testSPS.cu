#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostSkel.cu.h"

#ifndef INCLUDE_QUAD
#define INCLUDE_QUAD 0
#endif

// Initialize an array of int32_t with random values between -R and R.
// Array has length N.
// R seems to be max value of the elements of the array.
void initArrayInt32(int32_t* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i] = (rand() % M) - R;
    }
}

// Initialize an array of float with random values between -R and R.
// Array has length N.
// R seems to be max value of the elements of the array.
void initArrayFloat(float* inp_arr, const uint32_t N, const float R) {
    for(uint32_t i=0; i<N; i++) {
        float random_val = static_cast<float>(rand()) / static_cast<float>(RAND_MAX); // generates a random float between 0.0 and 1.0
        inp_arr[i] = 2.0f * R * random_val - R; // scales and shifts the random float to the range -R to R
    }
}

// Initialize an array of Quad<int32_t> with random values between -R and R.
// Array has length N.
// R seems to be max value of the elements of the array.
void initArrayQuadInt32(Quad<int32_t>* inp_arr, const uint32_t N, const int R) {
    const uint32_t M = 2*R+1;
    for(uint32_t i=0; i<N; i++) {
        inp_arr[i].x = (rand() % M) - R;
        inp_arr[i].y = (rand() % M) - R;
        inp_arr[i].z = (rand() % M) - R;
        inp_arr[i].w = (rand() % M) - R;
    }
}

int neq(int32_t a, int32_t b) {
	return a != b;
}

int neq(float a, float b) {
	printf("float version used\n");
	printf("a: %f, b: %f, diff: %f\n", a, b, fabs(a - b));
	return fabs(a - b) > 1.0e-4f;
}

int neq(Quad<int32_t> a, Quad<int32_t> b) {
	return a.x != b.x || a.y != b.y || a.z != b.z || a.w != b.w;
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */ 
template<typename T>
int bandwidthCudaMemcpy(const size_t N, T* d_in, T* d_out) {
    // dry run to exercise the d_out allocation!
    const size_t mem_size = N * sizeof(T);
	hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
			hipMemcpy(d_out, d_in, mem_size, hipMemcpyDeviceToDevice);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(T) * 1.0e-3f / elapsed;
        printf("- hipMemcpy runs in: %lu microsecs, GB/sec: %.2f\n",
			   elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );

    return 0;
}

/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 */
 template<typename T>
int bandwidthMemcpy(const size_t N, T* d_in, T* d_out) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + 1024 - 1) / 1024;
    naiveMemcpy<T><<< num_blocks, 1024>>>(d_out, d_in, N);
	hipDeviceSynchronize();

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<T><<< num_blocks, 1024 >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(T) * 1.0e-3f / elapsed;
        printf("- Naive Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );

    return 0;
}

/*
 * singlePassScanAuxBlock performs a single pass scan using an auxiliary block.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int singlePassScanAuxBlock(const size_t N, T* h_in,
	                       T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
    hipMemset(d_out, 0, N*sizeof(T));

    uint32_t num_blocks = (N+B*Q-1)/(B*Q) + 1;  // We add 1 to be our auxiliary block.
	size_t f_array_size = num_blocks - 1;
    int32_t* IDAddr;
    uint32_t* flagArr;
    T* aggrArr;
    T* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, -1, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(T));
    hipMemset(aggrArr, 0, f_array_size * sizeof(T));
    hipMalloc(&prefixArr, f_array_size * sizeof(T));
    hipMemset(prefixArr, 0, f_array_size * sizeof(T));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel1<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
	// time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, -1, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0, f_array_size * sizeof(T));
            hipMemset(prefixArr, 0, f_array_size * sizeof(T));
            SinglePassScanKernel1<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr);
            // printf("gpu %d\n", i + 1);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
        printf("- Single Pass Scan using aux block runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }
    gpuAssert( hipPeekAtLastError() );

    // The CPU we might as well just add the benchmark
    { // sequential computation
        T acc = T();
        for(uint32_t i=0; i<N; i++) {
            acc = h_in[i] + acc;
            h_ref[i] = acc;
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(neq(h_out[i], h_ref[i])) {
                printf("  - !!!INVALID!!!: Single Pass Scan at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("  - Single pass scan using aux block: VALID result!\n");
    }

	free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);

    return 0;
}

/*
 * singlePassScanLookback performs a single pass scan using lookback.
 * N - length of the input array
 * h_in - host input    of size: N * sizeof(int)
 * d_in - device input  of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int singlePassScanLookback(const size_t N, T* h_in,
                           T* d_in, T* d_out,
						   bool par_redux) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
	hipMemset(d_out, 0,  N *sizeof(T));

	uint32_t num_blocks = (N+B*Q-1)/(B*Q);
    size_t f_array_size = num_blocks;
    int32_t* IDAddr;
    uint32_t* flagArr;
    T* aggrArr;
    T* prefixArr;
    hipMalloc((void**)&IDAddr, sizeof(int32_t));
    hipMemset(IDAddr, 0, sizeof(int32_t));
    hipMalloc(&flagArr, f_array_size * sizeof(uint32_t));
    hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
    hipMalloc(&aggrArr, f_array_size * sizeof(T));
    hipMemset(aggrArr, 0, f_array_size * sizeof(T));
    hipMalloc(&prefixArr, f_array_size * sizeof(T));
    hipMemset(prefixArr, 0, f_array_size * sizeof(T));

    // dry run to exercise the d_out allocation!
    SinglePassScanKernel2<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr, par_redux);
    hipDeviceSynchronize();

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    // time the GPU computation
    // Need to reset the dynID and flag arr each time we call the kernel
    // Before we can start to run it multiple times and get a benchmark.
    {
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_GPU; i++) {
            hipMemset(IDAddr, 0, sizeof(int32_t));
            hipMemset(flagArr, X, f_array_size * sizeof(uint32_t));
            hipMemset(aggrArr, 0.0, f_array_size * sizeof(T));
            hipMemset(prefixArr, 0.0, f_array_size * sizeof(T));
            SinglePassScanKernel2<T><<< num_blocks, B>>>(d_in, d_out, N, IDAddr, flagArr, aggrArr, prefixArr, par_redux);
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
        elapsed = elapsed / RUNS_GPU;
        double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
		if (par_redux) {
        	printf("- Single Pass Scan using sequential lookback runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
		} else {
			printf("- Single Pass Scan using parallel lookback runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
		}
	}
    gpuAssert( hipPeekAtLastError() );

    // The CPU we might as well just add the benchmark
    { // sequential computation
        T acc = T();
        for(uint32_t i=0; i<N; i++) {
            acc = h_in[i] + acc;
            h_ref[i] = acc;
        }
        hipDeviceSynchronize();
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(neq(h_out[i], h_ref[i])) {
                printf("  - !!!INVALID!!!: Single Pass Scan at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
				printf("[");
				int resolution = 32;
				for (int k = -resolution; k < resolution; k++) {
					if (i + k >= 0) {
						printf("%d, ", h_out[i + k]);
					}
				}
				printf("]\n");
                exit(1);
            }
        }
        printf("  - Single pass scan using lookback: VALID result!\n");
    }

	free(h_out);
    free(h_ref);
    hipFree(IDAddr);
    hipFree(flagArr);
    hipFree(aggrArr);
    hipFree(prefixArr);

    return 0;
}

/*
 * cpuSeqScan computes a sequencial scan on the cpu.
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 * Returns 0 if the scan was successful, 1 otherwise.
 */
template<typename T>
int cpuSeqScan(const size_t N, T* h_in,
	           T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* h_out = (T*)malloc(mem_size);

    // dry run to exercise the h_out allocation!
    hipDeviceSynchronize();
    T acc = T();
    for(uint32_t i=0; i<N; i++) {
        acc = h_in[i] + acc;
        h_out[i] = acc;
    }
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    // The CPU we might as well just add the benchmark
    { // sequential computation
        gettimeofday(&t_start, NULL);
        for(int i=0; i<RUNS_CPU; i++) {
            T acc = T();
            for(uint32_t i=0; i<N; i++) {
                acc = h_in[i] + acc;
                h_out[i] = acc;
            }
        }
        hipDeviceSynchronize();
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_CPU;
        double gigaBytesPerSec = N * 2 * sizeof(T) * 1.0e-3f / elapsed;
        printf("- Scan Inclusive AddI32 CPU Sequential runs in: %lu microsecs, GB/sec: %.2f\n",
		       elapsed, gigaBytesPerSec);
    }

    free(h_out);
    return 0;
}

/*
 * scanIncAddI32 computes a scan inclusive add on the GPU using the
 * scanInc kernel we implemented in assignment 2.
 * b_size - desired CUDA block size ( <= 1024, multiple of 32)
 * N - length of the input array
 * h_in - host input of size: N * sizeof(int)
 * d_in - device input of size: N * sizeof(ElTp)
 * d_out - device result of size: N * sizeof(int)
 */
template<typename T>
int scanIncAdd(const uint32_t b_size, const size_t N, T* h_in,
				  T* d_in, T* d_out) {
    const size_t mem_size = N * sizeof(T);
    T* d_tmp;
    T* h_out = (T*)malloc(mem_size);
    T* h_ref = (T*)malloc(mem_size);
    hipMalloc((void**)&d_tmp, MAX_BLOCK*sizeof(T));
    hipMemset(d_out, 0, N*sizeof(T));

    // dry run to exercise d_tmp allocation
    scanInc< Add<T> > ( b_size, N, d_out, d_in, d_tmp );

    // time the GPU computation
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int i=0; i<RUNS_GPU; i++) {
        scanInc< Add<T> > ( b_size, N, d_out, d_in, d_tmp );
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
    double gigaBytesPerSec = N  * 2 * sizeof(T) * 1.0e-3f / elapsed;
    printf("- Scan Inclusive AddI32 GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"
          , elapsed, gigaBytesPerSec);

    gpuAssert( hipPeekAtLastError() );

    { // sequential computation
		T acc = T();
		for(uint32_t i=0; i<N; i++) {
			acc = acc + h_in[i];
			h_ref[i] = acc;
		}
    }

    { // Validation
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
        for(uint32_t i = 0; i<N; i++) {
            if(neq(h_out[i], h_ref[i])) {
                printf("  - !!!INVALID!!!: Scan Inclusive AddI32 at index %d, dev-val: %d, host-val: %d\n",
				       i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("  - Scan Inclusive AddI32: VALID result!\n");
    }

    free(h_out);
    free(h_ref);
    hipFree(d_tmp);

    return 0;
}

int i32Experiments(const uint32_t N) {
    const size_t mem_size = N*sizeof(int32_t);
    int32_t* h_in = (int32_t*) malloc(mem_size);
    int32_t* d_in;
    int32_t* d_out;
    hipMalloc((void**)&d_in ,   mem_size);
    hipMalloc((void**)&d_out,   mem_size);

	printf("Computing experiments with type: int32_t\n");
    initArrayInt32(h_in, N, 13);
	hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // Scan experiments.
    {
		// computing a "realistic/achievable" bandwidth figure
		bandwidthCudaMemcpy<int32_t>(N, d_in, d_out);
		bandwidthMemcpy<int32_t>(N, d_in, d_out);
		// Scan experiments.
        cpuSeqScan<int32_t>(N, h_in, d_in, d_out);
        singlePassScanAuxBlock<int32_t>(N, h_in, d_in, d_out);
        singlePassScanLookback<int32_t>(N, h_in, d_in, d_out, false);
        singlePassScanLookback<int32_t>(N, h_in, d_in, d_out, true);
		scanIncAdd<int32_t>(B, N, h_in, d_in, d_out);
    }

    // cleanup memory
    free(h_in);
    hipFree(d_in);
    hipFree(d_out);

	return 0;
}

#if INCLUDE_QUAD == 1
int quadInt32Experiments(const uint32_t N) {
	const size_t mem_size = N*sizeof(Quad<int32_t>);
	Quad<int32_t>* h_in    = (Quad<int32_t>*) malloc(mem_size);
	Quad<int32_t>* d_in;
	Quad<int32_t>* d_out;
	hipMalloc((void**)&d_in ,   mem_size);
	hipMalloc((void**)&d_out,   mem_size);

	printf("Computing experiments with type: Quad<int32_t>\n");
	initArrayQuadInt32(h_in, N, 13);
	hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

	// Scan experiments.
	{
		// computing a "realistic/achievable" bandwidth figure
		bandwidthCudaMemcpy<Quad<int32_t>>(N, d_in, d_out);
		bandwidthMemcpy<Quad<int32_t>>(N, d_in, d_out);
		// bandwidthGlgShrMemcpyInt32(N, h_in, d_in, d_out);
		// Scan experiments.
		cpuSeqScan<Quad<int32_t>>(N, h_in, d_in, d_out);
		singlePassScanAuxBlock<Quad<int32_t>>(N, h_in, d_in, d_out);
		singlePassScanLookback<Quad<int32_t>>(N, h_in, d_in, d_out, false);
		singlePassScanLookback<Quad<int32_t>>(N, h_in, d_in, d_out, true);
		scanIncAdd<Quad<int32_t>>(B, N, h_in, d_in, d_out);
	}

	// cleanup memory
	free(h_in);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
#endif


int main (int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array-length>\n", argv[0]);
        exit(1);
    }

    initHwd();

    const uint32_t N = atoi(argv[1]);

    printf("Testing parallel basic blocks for input length: %d and CUDA-block size: %d and Q: %d\n\n\n", N, B, Q);

	i32Experiments(N);
#if INCLUDE_QUAD == 1
	quadInt32Experiments(N);
#endif
	return 0;

}
